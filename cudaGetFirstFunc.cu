#include "hip/hip_runtime.h"
#include "getFirstFunc.h"

extern Mat src;

bool getFirst(Mat frame,Point* firstPoint)
{
		//---------------------------��򷨶�ֵ��ͼ��------------------------------
		const int rowDivision = 10,colDivision = 10;
		const int rowLength = src.rows/rowDivision,colLength = src.cols/colDivision;
		const int minimumGap = 50;

		for (int r = 0; r < rowDivision; r++)
		{
			for (int c = 0; c < colDivision; c++)
			{
				uchar* srcData = src.data;
				int otsuThreshold =	otsu(srcData,src.rows,src.cols,colLength*c,rowLength*r,colLength,rowLength,0,minimumGap);
				int beginRow = r*rowLength,beginCol = c*colLength;
				for (int rr = 0; rr < rowLength; rr++)
				{
					srcData = src.ptr<UCHAR>(beginRow+rr);
					srcData += beginCol;
					for(int cc = 0;cc<colLength;cc++)
					{
						if(*srcData > otsuThreshold)
							*srcData = 255;
						else
							*srcData = 0;
						srcData++;
					}
				}
			}
		}
		namedWindow("��򷨶�ֵ�ָ�",0);
		imshow("��򷨶�ֵ�ָ�",src);
		waitKey(1);
		//--------------------------------��򷨽���------------------------------------------------

	//------------------------canny���+hough�任----------------------------------------------------
	Canny(src, src, 50, 200,3);

	vector<Vec4i> lines;
	const int th = 30;
	const int thgap = 20;
	HoughLinesP(src, lines, 1, CV_PI/180, th, th, thgap );

	//���lines����һ������ôֱ���˳�
	//printf("line size: %d\n",lines.size());
	if(lines.size() > 1024)
		return false;
	//-----------------------canny���+hough�任����----------------------------------------------------

	//---------------------������ֱ����������չ---------------------------------
	int expandVal = 50;

	src.setTo(0);
	vector<Vec4i> concisedLines;
	Vec4i concisedPoint;
	
	for (int i=0;i<lines.size();i++){

		Vec4i l = (lines[i]);

		int x1 = l[0],y1 = l[1],x2 = l[2],y2 = l[3];

				//���ֱ��̫����ô��Ϊ���޹�ֱ�ߣ�ֱ��ȥ��
				int tooLong = src.cols / 4;
				if( pow(x1-x2,2)+pow(y1-y2,2) > pow(tooLong,2) )
					continue;

				//--------------����������չ-----------------

				int x1_x2 = x1-x2,y1_y2 = y1-y2;
				int gap = disCompute(x1,y1,x2,y2);
				float scale = float(expandVal)/float(gap);
				int x1new = int(scale*float(x1_x2) + x1);
				int y1new = int(scale*float(y1_y2) + y1);
				int x2new = int(scale*float(-1*x1_x2) + x2);
				int y2new = int(scale*float(-1*y1_y2) + y2);
				x1 = (max)(x1new,0);
				x1 = (min)(x1,src.cols-1);
				x2 = (max)(x2new,0);
				x2 = (min)(x2,src.cols-1);
				y1 = (max)(y1new,0);
				y1 = (min)(y1,src.rows-1);
				y2 = (max)(y2new,0);
				y2 = (min)(y2,src.rows-1);

				line(src,Point(x1,y1),Point(x2,y2),Scalar(255),3,CV_AA);

				concisedPoint[0] = x1;
				concisedPoint[1] = y1;
				concisedPoint[2] = x2;
				concisedPoint[3] = y2;
				concisedLines.push_back(concisedPoint);			  

	}
	//-------------------��չ��������------------------------------------

	//�ѽ������һ�£����ں�������ҵ��ĵ������ֱ��ͼ
	Mat srcCopy = src.clone();

	//-------------------���㽻��-------------------------------------------------------------------------------
	src.setTo(0);
	//�Ըղ��ҵ���ֱ�ߣ����»���ͼ�ϣ�����ʹ�õ��ӵķ�ʽ�����������Եõ���ֵ��
	for( size_t i = 0; i < concisedLines.size(); i++ )
	{
		Vec4i l = (concisedLines[i]);
		DrawLine(src,l[0],l[1],l[2],l[3],10,4);
	}
	namedWindow("��ֵͼ",0);
	imshow("��ֵͼ",src);
	imwrite("peakValue.jpg",src);
	waitKey(1);

	//����ֵ�㣬�洢maxListLength�����ĵ㣬������Щ�㣬�õ�����ֱ��ͼ������ǰ�ĸ���
	const int maxListLength = 1000;
	_maxList maxlist(maxListLength);

	for(int i=0;i<src.rows;i++)
	{
		uchar *pdata = src.ptr<uchar>(i);
		for(int j=0;j<src.cols;j++)
		{
			if(*pdata == 0){
				pdata ++;
				continue;
			}
			maxlist.insert(*pdata,i,j);
			pdata ++;
		}
	}
	maxlist.update(srcCopy,0.7);
	//------------------���㽻�����-------------------------------------------------

	if(maxlist.fgetPointNum() == 4){
		for (int i = 0; i < 4; i++){
			firstPoint[i] = maxlist.fgetpoint()[i];
		}
		return true;
	}
	else{
		return false;
	}
}