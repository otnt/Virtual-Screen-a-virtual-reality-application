#include "hip/hip_runtime.h"

#include "cudaWarpPerspective.h"

//һЩȫ�ֱ���
Mat pdfImg;
extern uchar* gpuDataSrc;
uchar* gpuDataImg;
float* gpuDataMapmap;
int* gpuDataRowAndCol;

bool InitCUDA()
{
    int count;

    hipGetDeviceCount(&count);
    if(count == 0) {
        fprintf(stderr, "There is no device./n");
        return false;
    }

    int i;
    for(i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if(prop.major >= 1) {
                break;
            }
        }
    }

    if(i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x./n");
        return false;
    }

    hipSetDevice(i);

    return true;
}

//data������Ҫ�洢��ͼƬ��imgData������ȥ��ͼƬ
__global__ void CudaWarpPerspectiveLeftHalf(uchar* srcData,uchar* imgData,float* Mdata,int*gpuDataRowAndCol,int imgChannels)//int srcRows,int srcCols,int imgRows,int imgCols)
{
	int tid = threadIdx.x;//tid��ʾ��col
	int bid = blockIdx.x;//bid��ʾ��row

				int newX = int((Mdata[0]*float(tid)+Mdata[1]*float(bid)+Mdata[2])/(Mdata[6]*float(tid)+Mdata[7]*float(bid)+Mdata[8]));
				int newY = int((Mdata[3]*float(tid)+Mdata[4]*float(bid)+Mdata[5])/(Mdata[6]*float(tid)+Mdata[7]*float(bid)+Mdata[8]));

				if(newX<0 || newX>gpuDataRowAndCol[3]-1 || newY<0 || newY>gpuDataRowAndCol[2]-1){
					return;
				}
				else{
					uchar *pdata = srcData + bid*gpuDataRowAndCol[1]*3 + tid*3;

					uchar *pdataImg  = imgData + newY*gpuDataRowAndCol[3]*imgChannels + newX*imgChannels;//  pdfImg.ptr<uchar>(newY);

					//�������ͨ����͸�����֣����Ƴ�
					if(imgChannels == 4 && *(pdataImg+3) == 0){
						return;
					}
					//pdataImg += 3*newX;
					*pdata++ = *pdataImg ++;
					*pdata++ = *pdataImg ++;
					*pdata = *pdataImg;
				}
}

__global__ void CudaWarpPerspectiveRightHalf(uchar* srcData,uchar* imgData,float* Mdata,int*gpuDataRowAndCol,int imgChannels)//int srcRows,int srcCols,int imgRows,int imgCols)
{
	int tid = threadIdx.x;//tid��ʾ��col
	tid += gpuDataRowAndCol[1]/2;
	int bid = blockIdx.x;//bid��ʾ��row

				int newX = int((Mdata[0]*float(tid)+Mdata[1]*float(bid)+Mdata[2])/(Mdata[6]*float(tid)+Mdata[7]*float(bid)+Mdata[8]));
				int newY = int((Mdata[3]*float(tid)+Mdata[4]*float(bid)+Mdata[5])/(Mdata[6]*float(tid)+Mdata[7]*float(bid)+Mdata[8]));

				if(newX<0 || newX>gpuDataRowAndCol[3]-1 || newY<0 || newY>gpuDataRowAndCol[2]-1){
					return;
				}
				else{
					uchar *pdata = srcData + bid*gpuDataRowAndCol[1]*3 + tid*3;

					uchar *pdataImg  = imgData + newY*gpuDataRowAndCol[3]*imgChannels + newX*imgChannels;//  pdfImg.ptr<uchar>(newY);

					//�������ͨ����͸�����֣����Ƴ�
					if(imgChannels == 4 && *(pdataImg+3) == 0){
						return;
					}
					//pdataImg += 3*newX;
					*pdata++ = *pdataImg ++;
					*pdata++ = *pdataImg ++;
					*pdata = *pdataImg;
				}
}

bool cudaCopyPdfOrVideoImg()
{
	hipMalloc((void**) &gpuDataImg, pdfImg.channels()*pdfImg.rows*pdfImg.cols*sizeof(uchar));
	hipMemcpy(gpuDataImg, pdfImg.data, pdfImg.channels()*pdfImg.rows*pdfImg.cols*sizeof(uchar), hipMemcpyHostToDevice);
	return true;
}

extern bool isVideo;
bool myCudaWarpInit(const char* pdfPicName,Mat src)
{
	if(!isVideo){
		//����ͼƬ
		pdfImg = imread(pdfPicName,-1);
		//��������
		//copy ������ȥ��ͼ
		hipMalloc((void**) &gpuDataImg, pdfImg.channels()*pdfImg.rows*pdfImg.cols*sizeof(uchar));
		hipMemcpy(gpuDataImg, pdfImg.data, pdfImg.channels()*pdfImg.rows*pdfImg.cols*sizeof(uchar), hipMemcpyHostToDevice);
	}
	//copy ����ͼ��row��col
	int cpuDataRowAndCol[4] = {src.rows,src.cols,pdfImg.rows,pdfImg.cols};
	hipMalloc((void**) &gpuDataRowAndCol,4*sizeof(int));
	hipMemcpy(gpuDataRowAndCol,cpuDataRowAndCol,4*sizeof(int),hipMemcpyHostToDevice);
	
	return true;
}

void myCudaRelease()
{
	hipFree(gpuDataImg);
	hipFree(gpuDataRowAndCol);
}

void cudaReleasePdfOrVideoImg()
{
	hipFree(gpuDataImg);
}

void myCudaWarp(Mat src,const Point* dest)
{
		const int NUM = src.rows*src.cols ;
		const int THREAD_NUM = src.cols/2;//cols�����������thread���ƣ�1024������/2
		const int BLOCK_NUM = src.rows;	

		//��Ӧ����
		CvPoint2D32f psrc[4],pdst[4];

		/*psrc[0].x = 0;psrc[0].y = 0;
		psrc[1].x = pdfImg.cols-1;psrc[1].y = 0;
		psrc[2].x = pdfImg.cols-1;psrc[2].y = pdfImg.rows-1;
		psrc[3].x = 0;psrc[3].y = pdfImg.rows-1;*/

		//��������һ�㣬����������ס�ĸ���־��
		//int offsetC = (double)src.rows*0.2143, offsetR = (double)src.rows*0.2143;
		int offsetC =pdfImg.cols*0.1, offsetR = pdfImg.rows*0.2;
		psrc[0].x = 0+offsetC;psrc[0].y = 0+offsetR;
		psrc[1].x = pdfImg.cols-1-offsetC;psrc[1].y = 0+offsetR;
		psrc[2].x = pdfImg.cols-1-offsetC;psrc[2].y = pdfImg.rows-1-offsetR;
		psrc[3].x = 0+offsetC;psrc[3].y = pdfImg.rows-1-offsetR;

		for (int i = 0; i < 4; i++)
		{
			pdst[i].x = dest[i].x;
			pdst[i].y = dest[i].y;
		}
		//����任����
		CvMat *mapmapcv = cvCreateMat(3,3,CV_32FC1);
			//��ô�dst��src��ӳ�䣬���Ǵ�src��dst��ӳ�䣡�����������ʡ��һ��invert
		cvGetPerspectiveTransform(pdst,psrc,mapmapcv);
		Mat mapmap = mapmapcv;
		float* pdatamap = (float*)mapmap.data;
			
		//copy �任����
		hipMalloc((void**) &gpuDataMapmap , 9*sizeof(float));
		hipMemcpy(gpuDataMapmap,pdatamap,9*sizeof(float),hipMemcpyHostToDevice);
		
		//����
		CudaWarpPerspectiveLeftHalf<<<BLOCK_NUM, THREAD_NUM>>>(gpuDataSrc,gpuDataImg,gpuDataMapmap,gpuDataRowAndCol,pdfImg.channels());
		CudaWarpPerspectiveRightHalf<<<BLOCK_NUM, THREAD_NUM>>>(gpuDataSrc,gpuDataImg,gpuDataMapmap,gpuDataRowAndCol,pdfImg.channels());

		//copy ����
		hipMemcpy(src.data, gpuDataSrc, NUM * 3 *sizeof(uchar), hipMemcpyDeviceToHost);

	return;
}

void cudaCopyFrame(Mat frame)
{
	//��������
	//copy Ҫ����ͼ�������ӵ�ͼ��
	hipMalloc((void**) &gpuDataSrc, frame.rows*frame.cols * frame.channels() *sizeof(uchar));
	hipMemcpy(gpuDataSrc, frame.data, frame.rows*frame.cols  * frame.channels() *sizeof(uchar), hipMemcpyHostToDevice);
}