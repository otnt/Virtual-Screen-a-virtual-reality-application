#include "hip/hip_runtime.h"

#include "cudaFindDarkestPoint.h"

extern uchar* gpuDataSrc;
int *gpuSearchSize, *gpuSearchSizeResult;

__global__ void getPatchValue(uchar*gpuSrcData,uchar*gpuSmallSeq,int sx,int sy,int rows,int cols,int subSize,int beginKernalSize)
{
	int tid = threadIdx.x;//tid��ʾ��col
	int bid = blockIdx.x;//bid��ʾ��row

	int px=sx-subSize/2 + tid;
	int py =sy-subSize/2 + bid;

	//��������֪ʶ����Ե����Բ�����⣬ֱ�ӷ���һ������ֵ
	if(py - beginKernalSize/2 <= 0 || py + beginKernalSize/2 >= rows-1
		|| px - beginKernalSize/2 <= 0 || px + beginKernalSize/2 >= cols-1){
		gpuSmallSeq[bid*subSize+tid]  = 0;
		return;
	}

	float result = 0.0;
	for(int i=-beginKernalSize/2;i<beginKernalSize/2;i++)
				{
					uchar* pdata_ori = gpuSrcData;
					pdata_ori += (py+i)*cols*3 + (px-beginKernalSize/2)*3;//src.ptr<uchar>(py+i);
					//pdata_ori += px-beginKernalSize/2;
					for(int j=-beginKernalSize/2;j<beginKernalSize/2;j++)
					{
						//Gray = 0.212671 * R + 0.715160 * G + 0.072169 * B +0 *A 
						result += (float)*pdata_ori * 0.072169;
						pdata_ori++;
						result += (float)*pdata_ori * 0.715160;
						pdata_ori++;
						result += (float)*pdata_ori * 0.212671;
						pdata_ori++;
					}
				}
	
	result = (double)(255*beginKernalSize*beginKernalSize - result)/(double)(beginKernalSize*beginKernalSize);
	gpuSmallSeq[bid*subSize+tid]  = (int)result;
	return;
}


//��src��sx��sy����subSize x subSize���ڣ�,Ѱ��beginKernalSize x beginKernalSize����ڵ����꣬����src�е�����
Point getDarkestPoint(Mat src,int sx,int sy,int subSize,int beginKernalSize,double ththreshold,double *histoGood)
{
	//----------------------ʹ��GPU������ЩbeginKernalSize x beginKernalSize�������ֵ֮�ͣ���һ����0~255��������ʾ��Ҳ����̫Ӱ�쾫��----------------------------
	const int NUM = subSize*subSize ;
	const int THREAD_NUM = subSize;
	const int BLOCK_NUM = subSize;	

	uchar *smallSeq1 = new uchar[subSize*subSize];
	uchar *gpuSmallSeq1;
	hipMalloc((void**) &gpuSmallSeq1, subSize*subSize*sizeof(uchar));
	getPatchValue<<<BLOCK_NUM, THREAD_NUM>>>(gpuDataSrc,gpuSmallSeq1,sx,sy,src.rows,src.cols,subSize,beginKernalSize);
	//copy ����
	hipMemcpy(smallSeq1, gpuSmallSeq1, subSize*subSize*sizeof(uchar), hipMemcpyDeviceToHost);
	//-----------------------------------GPU�������----------------------------------------------

	//------------------------------------��������ͼ����ڵĵ㣬��������ڵĵ�ȡλ��ƽ��---------------------------------
	//�����Լ����㷨���ð�T T
	int maxSmallSeq = 0;
	for(int i=0;i<subSize*subSize;i++){
		if(maxSmallSeq < smallSeq1[i])
			maxSmallSeq = smallSeq1[i];
	}
	int countSmallSeq = 0,rowrow = 0,colcol = 0;
	for(int i=0;i<subSize*subSize;i++){
		if(maxSmallSeq == smallSeq1[i]){
			countSmallSeq ++;
			rowrow += i/subSize;
			colcol += i%subSize;
		}
	}
	countSmallSeq = max(countSmallSeq,1);
	rowrow /= countSmallSeq;
	colcol /= countSmallSeq;
	//���ۣ�����
	swap(rowrow,colcol);
	//--------------------------------------------������------------------------------------------------------

	Mat forShowBlackCamShiftMat(subSize,subSize,CV_8UC1);
	forShowBlackCamShiftMat.data = (uchar*)smallSeq1;

	//-----------------------------------������ڵ�Ϊ���ĵ�histogram---------------------------------------
	//֮ǰ�ƺ���x����col����y����row
	int px = sx - subSize/2 + colcol, py = sy - subSize/2 + rowrow;
	int roiSize = subSize;//roi- roiSize * roiSize

	double *cpuHistogram1 = new double[256];
	memset(cpuHistogram1,0,256*sizeof(double));

	int countcount = 0;

	for(int i=-roiSize/2;i<roiSize/2;i++)
	{
		//py+i���ܻ�Խ��
		int tmp = py+i;
		if(tmp<0)
			continue;//ǰ���б�����ɫ
		else if (tmp > src.rows-1) 
			break;//����б�����ɫ

		uchar* pdata_ori = src.ptr<uchar>(tmp);

		//px-roiSize/2����С��0,�⼸��������������ɫ
		int continueCols = 0;
		if(px-roiSize/2 < 0) 
			continueCols = roiSize/2 -px;
		//px+roiSize/2����Խ�磬�⼸��Ҳ������ɫ
		int breakCols = roiSize;
		if(px+roiSize/2 > src.cols-1)
			breakCols = roiSize-((px+roiSize/2)-(src.cols -1));
		pdata_ori += (max)(px-roiSize/2,0);
		for(int j=-roiSize/2;j<roiSize/2;j++)
		{
			while(continueCols && continueCols--)//����ͷ����
			{
				pdata_ori++;
				continue;
			}
			if(breakCols-- == 0)//���������
				break;
			cpuHistogram1[*pdata_ori] ++;
			countcount ++;
			pdata_ori++;
		}
	}
	//ֱ��ͼ��һ��
	for (int i = 0; i < 256; i++){
		cpuHistogram1[i] /= countcount;
	}
	//--------------------------------------------��histogram����---------------------------------------------

	//--------------------------------histogramƥ��-----------------------------------------
	double match = 0;
	for (int i = 0; i < 256; i++){
		match += (min)(histoGood[i],cpuHistogram1[i]);
	}

	//���match����0.6�������ֱ��ͼ
	if(match >= 0.6){
		for (int i=0;i<256;i++){
			histoGood[i] = cpuHistogram1[i];
		}
	}

	circle(forShowBlackCamShiftMat,Point(rowrow,colcol),5,Scalar(0),-1,8,0);
	namedWindow("׷�ٵ�",0);
	imshow("׷�ٵ�",forShowBlackCamShiftMat);
	imwrite("׷�ٵ�.jpg",forShowBlackCamShiftMat);
	waitKey(1);
	//------------------------------ƥ�����--------------------------------------

	//-------------------------------���ص�----------------------------------------------------
	hipFree(gpuSmallSeq1);
	delete []smallSeq1;
	delete []cpuHistogram1;


	if(match > ththreshold)
	{
		return Point(sx - subSize/2 + rowrow, sy - subSize/2 + colcol);
	}
	else
		return Point(-1,-1);

}

//��src��sx��sy����subSize x subSize���ڣ�,Ѱ��beginKernalSize x beginKernalSize����ڵ����꣬����src�е�����
Point getFirstDarkestPoint(Mat src,int sx,int sy,int subSize,int beginKernalSize,double ththreshold,double* frameHistogram)
{
	//----------------------ʹ��GPU������ЩbeginKernalSize x beginKernalSize�������ֵ֮�ͣ���һ����0~255��������ʾ��Ҳ����̫Ӱ�쾫��----------------------------
	const int NUM = subSize*subSize ;
	const int THREAD_NUM = subSize;
	const int BLOCK_NUM = subSize;	

	uchar *smallSeq1 = new uchar[subSize*subSize];
	uchar *gpuSmallSeq1;
	hipMalloc((void**) &gpuSmallSeq1, subSize*subSize*sizeof(uchar));
	getPatchValue<<<BLOCK_NUM, THREAD_NUM>>>(gpuDataSrc,gpuSmallSeq1,sx,sy,src.rows,src.cols,subSize,beginKernalSize);
	//copy ����
	hipMemcpy(smallSeq1, gpuSmallSeq1, subSize*subSize*sizeof(uchar), hipMemcpyDeviceToHost);

	//--------------------------------------------������------------------------------------------------------

	//-------------------------------------�ҵ���ڵ�--------------------------------------------------
	//�����Լ����㷨���ð�T T
	int maxSmallSeq = 0;
	for(int i=0;i<subSize*subSize;i++){
		if(maxSmallSeq < smallSeq1[i])
			maxSmallSeq = smallSeq1[i];
	}
	int countSmallSeq = 0,rowrow = 0,colcol = 0;
	for(int i=0;i<subSize*subSize;i++){
		if(maxSmallSeq == smallSeq1[i]){
			countSmallSeq ++;
			rowrow += i/subSize;
			colcol += i%subSize;
		}
	}
	countSmallSeq = max(countSmallSeq,1);
	rowrow /= countSmallSeq;
	colcol /= countSmallSeq;
	//���ۣ�����
	swap(rowrow,colcol);

	Mat forShowBlackCamShiftMat(subSize,subSize,CV_8UC1);
	forShowBlackCamShiftMat.data = smallSeq1;
	//-----------------------------------�ҵ����------------------------------------------------

	//-----------------------------------���ֱ��ͼ--------------------------------------------
	//֮ǰ�ƺ���x����col����y����row
	int px = sx - subSize/2 + colcol, py = sy - subSize/2 + rowrow;
	int roiSize = subSize;//roi- roiSize * roiSize

	double *cpuHistogram1 = new double[256];
	memset(cpuHistogram1,0,256*sizeof(double));

	int countcount = 0;

	for(int i=-roiSize/2;i<roiSize/2;i++)
	{
		//py+i���ܻ�Խ��
		int tmp = py+i;
		if(tmp<0)
			continue;//ǰ���б�����ɫ
		else if (tmp > src.rows-1) 
			break;//����б�����ɫ

		uchar* pdata_ori = src.ptr<uchar>(tmp);

		//px-roiSize/2����С��0,�⼸��������������ɫ
		int continueCols = 0;
		if(px-roiSize/2 < 0) 
			continueCols = roiSize/2 -px;
		//px+roiSize/2����Խ�磬�⼸��Ҳ������ɫ
		int breakCols = roiSize;
		if(px+roiSize/2 > src.cols-1)
			breakCols = roiSize-((px+roiSize/2)-(src.cols -1));
		pdata_ori += (max)(px-roiSize/2,0);
		for(int j=-roiSize/2;j<roiSize/2;j++)
		{
			while(continueCols && continueCols--)//����ͷ����
			{
				pdata_ori++;
				//pdata_roi++;
				continue;
			}
			if(breakCols-- == 0)//���������
				break;
			cpuHistogram1[*pdata_ori] ++;
			countcount ++;
			//*pdata_roi = *pdata_ori;
			//pdata_roi++;
			pdata_ori++;
		}
	}
	//ֱ��ͼ��һ��
	for (int i = 0; i < 256; i++){
		cpuHistogram1[i] /= countcount;
	}

	//��һ֡�õ���ֱ��ͼ����Ϊ����ƥ��ı�׼
	for (int i = 0; i < 256; i++){
		frameHistogram[i] = cpuHistogram1[i];
	}

	circle(forShowBlackCamShiftMat,Point(rowrow,colcol),5,Scalar(0),-1,8,0);
	namedWindow("׷�ٵ�",0);
	imshow("׷�ٵ�",forShowBlackCamShiftMat);
	waitKey(1);
	//-------------------------------------ֱ��ͼ��ȡ����----------------------------------------------


	//------------------------------------------------------------------------------------------
	//���صĵ�
	Point returnPoint(sx - subSize/2 + rowrow, sy - subSize/2 + colcol);

	hipFree(gpuSmallSeq1);
	delete []smallSeq1;
	delete []cpuHistogram1;

	return returnPoint;
}

void cudaGetSearchSizeInit()
{
	hipMalloc((void**) &gpuSearchSize, 2*4*sizeof(int));//4���㣬ÿ������xy����
	hipMalloc((void**) &gpuSearchSizeResult,1*sizeof(int));
}

void cudaSearchSizeRelease()
{
	hipFree(gpuSearchSize);
	hipFree(gpuSearchSizeResult);
}

__global__ void cudaSearchSizeCompute(int* gpuPoints,int *result)
{
	int x1,y1,x2,y2,subsubSize;

	x1 = gpuPoints[0],y1 = gpuPoints[1];
	x2 = gpuPoints[2],y2 = gpuPoints[3];
	int disdis1 =  int(  sqrt(  float((x1-x2)*(x1-x2))+float((y1-y2)*(y1-y2))  )  );

	x1 = gpuPoints[2],y1 = gpuPoints[3];
	x2 = gpuPoints[4],y2 = gpuPoints[5];
	int disdis2 =  int(  sqrt(  float((x1-x2)*(x1-x2))+float((y1-y2)*(y1-y2))  )  );

	subsubSize = min(disdis1,disdis2);
	subsubSize = (double)(subsubSize)*0.3214;//���ڲ����õ��Ľ��
	subsubSize = max(1,subsubSize);

	*result = subsubSize;
}

void cudaGetSearchSizeCompute(Point* p)
{
	int cpuPoints[8] = 
	{
		p[0].x,p[0].y,
		p[1].x,p[1].y,
		p[2].x,p[2].y,
		p[3].x,p[3].y
	};
	hipMemcpy(gpuSearchSize,cpuPoints,2*4*sizeof(int),hipMemcpyHostToDevice);
	cudaSearchSizeCompute<<<1,1>>>(gpuSearchSize,gpuSearchSizeResult);
}

int cudaSearchSizeRetrive()
{
	int cpuSearchSizeResult;
	hipMemcpy(&cpuSearchSizeResult,gpuSearchSizeResult,1*sizeof(int),hipMemcpyDeviceToHost);
	return cpuSearchSizeResult;
}
